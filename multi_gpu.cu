#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define B 64
#define threadNum 32

const int INF = 1000000000;
void input(char *inFileName);
void output(char *outFileName);

void block_FW();
int ceil(int a, int b);
__global__ void phase1(int* dist, int Round, int n, size_t pitch);
__global__ void phase2(int* dist, int Round, int start_block, int end_block, int n, size_t pitch);
__global__ void phase3(int* dist, int Round, int start_block, int end_block, int n, size_t pitch);

int n, m;	
int *Dist = NULL;
int *device_Dist[2] = {NULL, NULL};
size_t pitch[2];

int main(int argc, char* argv[]) {
	omp_set_num_threads(2);
	input(argv[1]);
	block_FW();
	output(argv[2]);
	hipHostFree(Dist);
	return 0;
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

	hipHostMalloc(&Dist, (size_t)n*n*sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < n; ++ i) {
        for (int j = 0; j < n; ++ j) {
			Dist[i*n+j] = (i==j) ? 0 : INF;
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++ i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*n+pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char *outFileName) {
	FILE *outfile = fopen(outFileName, "wb");
	fwrite(Dist, sizeof(int), n*n, outfile);	
    fclose(outfile);
}

int ceil(int a, int b) {
	return (a + b - 1) / b;
}

void block_FW() {
	unsigned int round = ceil(n, B);
	dim3 block_p1 = {1, 1};
	dim3 block_p2 = {2, round};
	dim3 block_p3 = {round, round};
	dim3 threads = {threadNum, threadNum};
	int start_block[2] = {0, (int)round/2}, end_block[2] = {(int)round/2, (int)round}, offset[2]={((int)round/2)*B, n-((int)round/2)*B};
	#pragma omp parallel
	{
		int id = omp_get_thread_num();
		hipSetDevice(id);
		hipMallocPitch(&device_Dist[id], &pitch[id], (size_t)n*sizeof(int), (size_t)n);
		hipMemcpy2D((int*)((char*)device_Dist[id]+start_block[id]*B*pitch[id]), pitch[id], Dist+start_block[id]*B*n, (size_t)n*sizeof(int), (size_t)n*sizeof(int), (size_t)offset[id], hipMemcpyHostToDevice);
		for (unsigned int r = 0; r < round; ++r) {
			int cur = (r>=start_block[id] && r<end_block[id]) ? 1 : 0;
			int height = (r<round-1) ? B : n-r*B, width = height;
			if(cur){
				phase1<<<block_p1, threads>>>(device_Dist[id], r, n, pitch[id]);
				hipMemcpy2D(Dist+r*B*n+r*B, (size_t)n*sizeof(int), (int*)((char*)device_Dist[id]+r*B*pitch[id])+r*B, pitch[id], (size_t)width*sizeof(int), (size_t)height, hipMemcpyDeviceToHost);		
			}
			#pragma omp barrier
			if(!cur){
				hipMemcpy2D((int*)((char*)device_Dist[id]+r*B*pitch[id])+r*B, pitch[id], Dist+r*B*n+r*B, (size_t)n*sizeof(int), (size_t)width*sizeof(int), (size_t)height, hipMemcpyHostToDevice);
			}
			phase2<<<block_p2, threads>>>(device_Dist[id], r, start_block[id], end_block[id], n, pitch[id]);
			if(cur){
				hipMemcpy2D(Dist+r*B*n, (size_t)n*sizeof(int), (int*)((char*)device_Dist[id]+r*B*pitch[id]), pitch[id], (size_t)n*sizeof(int), (size_t)height, hipMemcpyDeviceToHost);		
			}
			#pragma omp barrier
			if(!cur){
				hipMemcpy2D((int*)((char*)device_Dist[id]+r*B*pitch[id]), pitch[id], Dist+r*B*n, (size_t)n*sizeof(int), (size_t)n*sizeof(int), (size_t)height, hipMemcpyHostToDevice);
			}
			phase3<<<block_p3, threads>>>(device_Dist[id], r, start_block[id], end_block[id], n, pitch[id]);
		}
		hipMemcpy2D(Dist+start_block[id]*B*n, (size_t)n*sizeof(int), (int*)((char*)device_Dist[id]+start_block[id]*B*pitch[id]), pitch[id], (size_t)n*sizeof(int), (size_t)offset[id], hipMemcpyDeviceToHost);
		hipFree(device_Dist[id]);
	}
}

__global__ void phase1(int* dist, int Round, int n, size_t pitch){

	int base = Round*B;
	int shift = B/threadNum;
	int i_st = base + threadIdx.x*shift, i_ed = i_st + shift;
	int j_st = base + threadIdx.y*shift, j_ed = j_st + shift;

	if(i_ed > n){
		i_ed = n;
	}
	if(j_ed > n){
		j_ed = n;
	}

	__shared__ int sm[B][B];

	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			int *dij = (int*)((char*)dist+pitch*i)+j;
			sm[i-base][j-base] = *dij;
		}
	}
	__syncthreads();

	int len = ((Round+1)*B < n) ? B : n - (Round)*B;

	#pragma unroll
	for (int k = 0; k < len; ++k) {
		#pragma unroll
		for(int i = i_st; i<i_ed ; ++i){
			#pragma unroll
			for(int j = j_st ; j<j_ed ; ++j){
				int relax = sm[i-base][k] + sm[k][j-base];
				if(relax < sm[i-base][j-base]){
					sm[i-base][j-base] = relax;
				}
			}
		}
		__syncthreads();
	}

	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			int *dij = (int*)((char*)dist+pitch*i)+j;
			*dij = sm[i-base][j-base];
		}
	}
}

__global__ void phase2(int* dist, int Round, int start_block, int end_block, int n, size_t pitch){
	if(blockIdx.y==Round)
		return;

	__shared__ int sm[2][B][B];
	
	int base_i = (1-blockIdx.x)*Round*B + blockIdx.x*blockIdx.y*B;
	int base_j = blockIdx.x*Round*B + (1-blockIdx.x)*blockIdx.y*B;
	int shift = B/threadNum;
	int i_st = base_i + threadIdx.x*shift, i_ed = i_st + shift; 
	int j_st = base_j + threadIdx.y*shift, j_ed = j_st + shift;	

	if(i_ed < B*start_block || i_st >= B*end_block)
		return;
	
	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			if(i<n && j<n){
				int *dij = (int*)((char*)dist+pitch*i)+j;
				sm[0][i-base_i][j-base_j] = *dij;
			}
			if(Round*B+(i-base_i)<n && Round*B+(j-base_j)<n){
				int *dkk = (int*)((char*)dist+pitch*(Round*B+(i-base_i))) + Round*B+(j-base_j);
				sm[1][i-base_i][j-base_j] = *dkk;
			}
		}
	}
	__syncthreads();

	if(i_ed > n){
		i_ed = n;
	}
	if(j_ed > n){
		j_ed = n;
	}
	int len = ((Round+1)*B < n) ? B : n - (Round)*B;
	int i_offset = i_st-base_i, i_len = i_ed - i_st;
	int j_offset = j_st-base_j, j_len = j_ed - j_st;
	
	#pragma unroll
	for(int i=i_offset ; i<i_offset+i_len ; ++i){
		#pragma unroll
		for(int j=j_offset ; j<j_offset+j_len ; ++j){
			#pragma unroll
			for (int k = 0; k < len; ++k) {
				int relax = sm[1-blockIdx.x][i][k] + sm[blockIdx.x][k][j];
				if(relax < sm[0][i][j]){
					sm[0][i][j] = relax;
				}
			}
			int *dij = (int*)((char*)dist+pitch*(base_i+i))+base_j+j;
			*dij = sm[0][i][j];
		}
	}
}

__global__ void phase3(int* dist, int Round, int start_block, int end_block, int n, size_t pitch){
	if(blockIdx.x==Round || blockIdx.y==Round)
		return;

	if(blockIdx.x < start_block || blockIdx.x >= end_block)
		return;

	__shared__ int sm[2][B][B];
	
	int base_i = blockIdx.x*B;
	int base_j = blockIdx.y*B;
	int shift = B/threadNum;
	int i_st = base_i + threadIdx.x*shift, i_ed = i_st + shift;
	int j_st = base_j + threadIdx.y*shift, j_ed = j_st + shift;
	
	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			if(i<n && Round*B+(j-base_j)<n){
				int *dik = (int*)((char*)dist+pitch*i)+Round*B+(j-base_j);
				sm[0][j-base_j][i-base_i] = *dik;
			}
			if(Round*B+(i-base_i)<n && j<n){
				int *dkj = (int*)((char*)dist+pitch*(Round*B+(i-base_i)))+j;
				sm[1][i-base_i][j-base_j] = *dkj;
			}
		}
	}
	__syncthreads();
	
	if(i_ed > n){
		i_ed = n;
	}
	if(j_ed > n){
		j_ed = n;
	}
	int len = ((Round+1)*B < n) ? B : n - (Round)*B;
	int i_offset = i_st-base_i, i_len = i_ed - i_st;
	int j_offset = j_st-base_j, j_len = j_ed - j_st;
	
	#pragma unroll
	for(int i = 0 ; i < i_len ; ++i){
		#pragma unroll
		for(int j= 0 ; j < j_len ; ++j){
			int *dij = (int*)((char*)dist+pitch*(i_st+i))+j_st+j;
			int ans = *dij;
			#pragma unroll
			for (int k = 0; k < len; ++k) {
				int relax = sm[0][k][i_offset+i] + sm[1][k][j_offset+j];
				if(relax < ans){
					ans = relax;
				}
			}
			*dij = ans;
		}
	}
}